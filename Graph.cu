#include "hip/hip_runtime.h"
#include <iostream>
#include "func.cuh"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <ctime>

const int layer = 3;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 2, 2, 784 }, Wnum = 0, Onum = 0, Dnum = 0, dop = 0;
	float* del, * delw, * weight, * out, * Inp, * Oout, pixel = 0;
	clock_t t1;
	cv::Mat result(28, 28, CV_8UC1);

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum << std::endl;

	float* weights = new float[WeightSum];
	float* InputDataArr = new float[n[0]];
	float* outO = new float[n[layer - 1]];

	for (int i = 0; i < n[layer - 1]; i++)
		outO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * sizeof(float));

	WeightGen << <WeightSum, 1 >> > (weight, WeightSum);
	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	InputDataArr[0] = 0.524;
	InputDataArr[1] = 0.524;
	hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
	InputData << <n[0], 1 >> > (Inp, out, n[0]);

	t1 = clock();
	for (int ad = 0; ad < 1; ad++) {
		for (int num = 0; num < 5000; num++) {
			for (int k = 0; k < 2; k++) {
				cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k * 6) + "\\" + std::to_string(1 + k * 12) + ".png");
				for (int i = 0; i < 28; i++) {
					for (int j = 0; j < 28; j++) {
						float per = 0;
						per = image.at<cv::Vec3b>(i, j)[0];
						per = per / 255;
						outO[i * 28 + j] = per;
					}
				}
				hipMemcpy(Oout, outO, n[layer - 1] * sizeof(float), hipMemcpyHostToDevice);
				
				Iteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
				
				Onum = NeuralSum - n[layer - 1];
				hipMemcpy(weights, out, NeuralSum * sizeof(float), hipMemcpyDeviceToHost);
				for (int i = 0; i < 28; i++) {
					for (int j = 0; j < 28; j++) {
						float per = 0;
						per = weights[Onum + i * 28 + j];
						per = ceil(per * 255);
						//std::cout << per << std::endl;
						result.at<uchar>(i, j) = per;
					}
				}
				cv::imshow("admin", result);
				cv::waitKey(1000);
				Onum = 0;
			}
		}

	}

	std::cout << "Time " << clock() - t1 << std::endl;

	for (int i = 0; i < 100; i++) {
		std::cin >> InputDataArr[0];
		hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
		InputData << <n[0], 1 >> > (Inp, out, n[0]);
		for (int i = 0; i < (layer - 1); i++) {
			Sumfunc << <n[i + 1], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);										//int layer, int Wnum, int Onum, float* weight, float* out
			Wnum = Wnum + n[i] * n[i + 1];
			Onum = Onum + n[i];
		}

		Onum = NeuralSum - n[layer - 1];
		hipMemcpy(weights, out, NeuralSum * sizeof(float), hipMemcpyDeviceToHost);
		for (int i = 0; i < 28; i++) {
			for (int j = 0; j < 28; j++) {
				float per = 0;
				per = weights[Onum + i * 28 + j];
				per = ceil(per * 255);
				result.at<uchar>(i, j) = per;
			}
		}
		cv::imshow("admin", result);
		cv::waitKey(1);
		Onum = 0;
		Wnum = 0;
	}
}
