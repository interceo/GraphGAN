#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <ctime>

#include "func.cuh"

const int layer = 3;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 784, 15, 10 /*enter*/}, Wnum = 0, Onum = 0, Dnum = 0, dop = 0;
	float* del, * delw, * weight, * out, * Inp, * Oout, pixel = 0, max = 10, min = -10;
	clock_t t1;

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum << std::endl;

	float* weights = new float[NeuralSum];
	float* InputDataArr = new float[n[0]];
	float* outO = new float[n[layer - 1]];

	for (int i = 0; i < n[layer - 1]; i++)
		outO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * sizeof(float));

	WeightGen << <WeightSum, 1 >> > (weight, WeightSum);
	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	//cv::Mat image = cv::imread("E:\\1.jpg");
	cv::Mat result(1200, 1200, CV_8UC1);
	cv::Mat result(image.rows, image.cols, CV_8UC1);

	t1 = clock();
	for (int num = 0; num < 500; num++) {
		for (int k = 0; k < 10; k++) {
			cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num) + ").png");
			OutputData(n, layer, outO, Oout, image, result, InputDataArr, Inp, out);
			NumberInp(out, image, InputDataArr, n, layer);
			Iteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
			Out(NeuralSum, layer, n, weights, out, image, result);
		}
	}

	std::cout << "Time " << clock() - t1 << std::endl;

	for (int i = 0; i < 100; i++) {
		//InputDataArr[0] = 0.524;
		std::cin >> InputDataArr[0];
		hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
		InputData << <n[0], 1 >> > (Inp, out, n[0]);
		for (int i = 0; i < (layer - 1); i++) {
			Sumfunc << <n[i + 1], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);										//int layer, int Wnum, int Onum, float* weight, float* out
			Wnum = Wnum + n[i] * n[i + 1];
			Onum = Onum + n[i];
		}

		Out(NeuralSum, layer, n, weights, out, result, result);
		Onum = 0;
		Wnum = 0;
	}
}
