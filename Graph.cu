#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <fstream>
#include <ctime>

#include "func.cuh"

const int layer = 4, RGB = 1;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 1, 2, 4, 784 }, int dop = 0,
		DisWeightSum = 0, DisNeuralSum = 0, nc[layer] = { 784, 64, 16, 1 };
	float * del, * delw, * weight, * out, * Inp, * Oout,
		  * Disdel, * Disdelw, * Disweight, * Disout, * DisInp, * DisOout;
	clock_t t1;
	std::string filename;

	/*---------перевести в const---------*/
	/*определение основных переменных Gen*/

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum * RGB << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum * RGB << std::endl;

	for (int i = 0; i < layer; i++)
		DisNeuralSum = nc[i] + DisNeuralSum;
	std::cout << "Neurals: " << DisNeuralSum * RGB << std::endl;

	for (int i = 0; i < layer - 1; i++)
		DisWeightSum = nc[i] * nc[i + 1] + DisWeightSum;
	std::cout << "Weights: " << DisWeightSum * RGB << std::endl;
	std::cout << std::endl;

	/*---------перевести в const---------*/

	float* outO = new float[n[layer - 1] * RGB];                //оптимальный вариант генератора в RAM
	float* DisoutO = new float[nc[layer - 1] * RGB];            //оптимальный вариант дискриминатора в RAM
	float* DisResult = new float[nc[layer - 1] * RGB];          //решение дискриминатора

	for (int i = 0; i < n[layer - 1] * RGB; i++)
		outO[i] = 0;
	for (int i = 0; i < nc[layer - 1] * RGB; i++)
		DisoutO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * RGB * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * RGB * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * RGB * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * RGB * sizeof(float));                              //входные данные генератора в GPU RAM
	hipMalloc((void**)&Oout, n[layer - 1] * RGB * sizeof(float));                     //оптимальный вариант генератора в GPU RAM

	hipMalloc((void**)&Disout, DisNeuralSum * RGB * sizeof(float));
	hipMalloc((void**)&Disdel, (DisNeuralSum - nc[0]) * sizeof(float));
	hipMalloc((void**)&Disweight, DisWeightSum * RGB * sizeof(float));
	hipMalloc((void**)&Disdelw, DisWeightSum * RGB * sizeof(float));
	hipMalloc((void**)&DisInp, nc[0] * RGB * sizeof(float));                          //входные данные дискриминатора в GPU RAM
	hipMalloc((void**)&DisOout, nc[layer - 1] * RGB * sizeof(float));                 //оптимальный вариант дискриминатора в GPU RAM

	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);
	DelwNull << < DisWeightSum, 1 >> > (Disdelw, DisWeightSum);

	cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\0\\1 (9).png");
	cv::Mat result(image.rows, image.cols, CV_8UC1);

	//InputOutImage(n[layer - 1], outO, Oout, image, RGB);
	DataCheck(WeightSum, weight, delw, 0, RGB);
	DataCheck(DisWeightSum, Disweight, Disdelw, 1, RGB);

	t1 = clock();
	for (int adm = 0; adm < 1000; adm++) {
		std::cout << "Iter #" << adm + 1 << std::endl;
		for (int l = 0; l < 1; l++) {
			for (int num = 0; num < 1; num++) {
				for (int k = 0; k < 5000; k++) {
					cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(1) + "\\1 (" + std::to_string(k + 1) + ").png");
					OptDis(DisoutO, nc[layer - 1], RGB, DisOout, 1);

					InputInputImage(nc[0], Disout, DisoutO, DisInp, image, DisNeuralSum, RGB);
					GlobalSumFunc(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, RGB);
					ImageResult(DisNeuralSum, Disout, nc[layer - 1], RGB);
					GenIteration(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, Disdelw, DisOout, DisoutO, Disdel, RGB);

					Random(nc[0], DisNeuralSum, DisInp, Disout, RGB);
					OptDis(DisoutO, nc[layer - 1], RGB, DisOout, 0);
					GlobalSumFunc(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, RGB);
					DisIteration(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, Disdelw, DisOout, DisoutO, Disdel, RGB);
					ImageResult(DisNeuralSum, Disout, nc[layer - 1], RGB);
					//std::cout << "12";
					//ImageResult(DisNeuralSum, Disout, nc[layer - 1], RGB);
				}
				dop = 1;
				//std::cout << "12";

				for (int k = 0; k < 1; k++) {
					cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(0) + "\\1 (" + std::to_string(k + 1) + ").png");
					/*Input(n[layer - 1], outO, Oout, image);*/
					// Random(n[0], NeuralSum, Inp, out);
					/*InputDiffer(dop, n, layer, WeightSum, nc[0], DisNeuralSum, NeuralSum, DisInp, Disout, weight, out, outO, dop, Inp);
					GlobalSumFunc(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout);
					ImageOpt(DisNeuralSum, Disout, nc[layer - 1], outO);*/
					//InputOutImage(n[layer - 1], outO, Oout, image);
					//Random(n[0], NeuralSum, Inp, out, RGB);
					Random(n[0], NeuralSum, Inp, out, RGB);
					//GlobalSumFunc(n, layer, NeuralSum, WeightSum, weight, out, RGB);
					//InputData << <n[0], 1 >> > (Inp, out, n[0], 0, 0);
					InputOutImage(n[layer - 1], outO, Oout, image, RGB);
					GlobalSumFunc(n, layer, NeuralSum, WeightSum, weight, out, RGB);
					GenIteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del, RGB);
					Out(NeuralSum, layer, n, weight, out, result);
					//OutOutImage(NeuralSum, layer, n, out, result, RGB);
					//std::cout << WeightSum << std::endl;
					//cv::waitKey(500);
				}
			}
		}
		Backup(WeightSum, weight, delw, 0, RGB);
		Backup(DisWeightSum, Disweight, Disdelw, 1, RGB);
	}
}
