#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <fstream>
#include <ctime>

#include "func.cuh"

const int layer = 3;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 16, 64, 784 }, int dop = 0,
		DisWeightSum = 0, DisNeuralSum = 0, nc[layer] = {784, 64, 1 };
	float * del, * delw, * weight, * out, * Inp, * Oout,
		  * Disdel, * Disdelw, * Disweight, * Disout, * DisInp, * DisOout;
	clock_t t1;
	std::string filename;

	/*---------перевести в const---------*/
	/*определение основных переменных Gen*/

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum * 3 << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum * 3 << std::endl;
	std::cout << std::endl;

	/*---------перевести в const---------*/

	float* outO = new float[n[layer - 1] * 3];
	float* DisoutO = new float[nc[layer - 1] * 3];

	for (int i = 0; i < n[layer - 1] * 3; i++)
		outO[i] = 0;
	for (int i = 0; i < nc[layer - 1] * 3; i++)
		DisoutO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * 3 * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * 3 * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * 3 * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * 3 * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * 3 * sizeof(float));

	hipMalloc((void**)&Disout, NeuralSum * 3 * sizeof(float));
	hipMalloc((void**)&Disdel, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&Disweight, WeightSum * 3 * sizeof(float));
	hipMalloc((void**)&Disdelw, WeightSum * 3 * sizeof(float));
	hipMalloc((void**)&DisInp, n[0] * 3 * sizeof(float));
	hipMalloc((void**)&DisOout, n[layer - 1] * 3 * sizeof(float));

	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\0\\1 (9).png");
	cv::Mat result(image.rows, image.cols, CV_8UC3);
	InputOutImage(n[layer - 1], outO, Oout, image);
	DataCheck(WeightSum, weight, delw, 0);
	//DataCheck(DisWeightSum, Disweight, Disdelw, 1);

	t1 = clock();
	for (int adm = 0; adm < 1000; adm++) {
		std::cout << "Iter #" << adm + 1 << std::endl;
		for (int l = 0; l < 1; l++) {
			for (int num = 0; num < 5000; num++) {
				for (int k = 0; k < 1; k++) {
					OutDiffer(dop, nc[0], DisNeuralSum, DisInp, Disout, DisoutO, image, 0);
				}

				for (int k = 0; k < 1; k++) {
					/*cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num + 1) + ").png");
					Input(n[layer - 1], outO, Oout, image);*/
					Random(n[0], NeuralSum, Inp, out, 0);
					GlobalSumFunc(n, layer, NeuralSum, WeightSum, weight, out);
					Iteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
					OutOutImage(NeuralSum, layer, n, out, result);
					//std::cout << WeightSum << std::endl;
				}
			}
		}
		Backup(WeightSum, weight, delw, 0);
	}
}
