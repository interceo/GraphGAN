#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <fstream>
#include <ctime>

#include "func.cuh"

const int layer = 3;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 16, 64, 784 };
	float* del, * delw, * weight, * out, * Inp, * Oout;
	clock_t t1;
	std::string filename;

	/*---------перевести в const---------*/
	/*определение основных переменных Gen*/

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum * 3 << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum * 3 << std::endl;
	std::cout << std::endl;

	/*---------перевести в const---------*/

	float* outO = new float[n[layer - 1] * 3];

	for (int i = 0; i < n[layer - 1] * 3; i++)
		outO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * 3 * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * 3 * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * 3 * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * 3 * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * 3 * sizeof(float));

	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\0\\1 (9).png");
	cv::Mat result(image.rows, image.cols, CV_8UC3);
	InputOutImage(n[layer - 1], outO, Oout, image);
	DataCheck(WeightSum, weight, delw, 0);

	t1 = clock();
	for (int adm = 0; adm < 1000; adm++) {
		std::cout << "Iter #" << adm + 1 << std::endl;
		for (int l = 0; l < 1; l++) {
			for (int num = 0; num < 5000; num++) {
				for (int k = 0; k < 1; k++) {
					/*cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num + 1) + ").png");
					Input(n[layer - 1], outO, Oout, image);*/
					Random(n[0], NeuralSum, Inp, out, 0);
					GlobalSumFunc(n, layer, NeuralSum, WeightSum, weight, out);
					Iteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
					OutOutImage(NeuralSum, layer, n, out, result);
				}
			}
		}
		Backup(WeightSum, weight, delw, 0);
	}
}
