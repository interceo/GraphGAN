#include "hip/hip_runtime.h"
#include <windows.h>
#include <iostream>
#include <cmath>
#include "func.cuh"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <ctime>
#include <fstream>
#include <cstring>

using namespace std;
using namespace cv;

const int coat = 3;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[coat] = { 784, 400, 10 }, Wnum = 0, Onum = 0, Dnum = 0, dop = 0;
	float* del, * delw, * weight, * out, * Inp, * Oout;
	float pixel = 0;
	clock_t t1, t2;
	string name = "E:\\Foton\\ngnl_data\\training\\", filename;

	for (int i = 0; i < coat; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum << std::endl;

	for (int i = 0; i < coat - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum << std::endl;

	float* weights = new float[WeightSum];
	float* InputDataArr = new float[n[0]];
	float* outO = new float[n[coat - 1]];
	
	for (int i = 0; i < n[coat - 1]; i++)
		outO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * sizeof(float));
	hipMalloc((void**)&Oout, n[coat - 1] * sizeof(float));

	WeightCreation << <WeightSum, 1 >> > (weight, WeightSum);
	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	t1 = clock();
	for (int ad = 0; ad < 10; ad++) {
		for (int num = 0; num < 10; num++) {
			ifstream nam(name + to_string(num) + ".txt");
			for (int k = 0; k < 2; k++) {
				nam >> filename;
				outO[num] = 1;
				Mat image = imread(name + to_string(num) + "\\" + filename);
				for (int i = 0; i < image.cols; i++) {
					for (int j = 0; j < image.rows; j++) {
						for (int p = 0; p < 3; p++) {
							pixel = pixel + image.at<Vec3b>(i, j)[p];
						}
						InputDataArr[i * image.rows + j] = pixel / 765;
						pixel = 0;
					}
				}

				hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(Oout, outO, n[coat - 1] * sizeof(float), hipMemcpyHostToDevice);
				InputData << <n[0], 1 >> > (Inp, out, n[0]);

				for (int i = 0; i < (coat - 1); i++) {
					Sumfunc << <n[i + 1], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);										//int coat, int Wnum, int Onum, float* weight, float* out
					Wnum = Wnum + n[i] * n[i + 1];
					Onum = Onum + n[i];
				}

				Onum = NeuralSum - n[coat - 1];
				Delta << <n[coat - 1], 1 >> > (Oout, out, del, Onum, n[coat - 1]);
				Wnum = WeightSum;

				for (int j = 0; j < coat - 1; j++) {
					Onum = Onum - n[coat - 2 - j];
					Wnum = Wnum - n[coat - 2 - j] * n[coat - 1 - j];
					DeltaN << <n[coat - 2 - j], 1 >> > (Dnum, Wnum, Onum, del, weight, out, n[coat - 1 - j], n[coat - 2 - j]);					    //int Dnum, int Wnum, int Onum, float* del, float* weight, float* out
					Dnum = Dnum + n[coat - 1 - j];
				}

				Wnum = WeightSum;
				Dnum = 0;
				Onum = NeuralSum - n[coat - 1];

				for (int j = 0; j < coat - 1; j++) {
					Onum = Onum - n[coat - 2 - j];
					Wnum = Wnum - n[coat - 1 - j] * n[coat - 2 - j];
					Deltaw << < n[coat - 2 - j], 1 >> > (weight, del, out, delw, Dnum, Onum, Wnum, n[coat - 1 - j], n[coat - 2 - j]);				//float* weight, float* del, float* out, float* delw, int Dnum, int Onum, int Wnum, int coat, int n
					Dnum = Dnum + n[coat - 1 - j];
				}

				Wnum = 0;
				Onum = 0;
				Dnum = 0;
				outO[num] = 0;

				hipMemcpy(weights, out, (NeuralSum) * sizeof(float), hipMemcpyDeviceToHost);
				for (int i = (NeuralSum - n[coat - 1]); i < NeuralSum; i++) {
					//std::cout << InputDataArr[0] << "" << InputDataArr[1] << " ";
					cout << weights[i] /*<< " " << outO[0]*/ << endl;;
				}
			}
			cout << endl;
		}
	}
	std::cout << "Time: " << clock() - t1 << std::endl;
}
