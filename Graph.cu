#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <fstream>
#include <ctime>

#include "func.cuh"

const int layer = 5;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 2, 32, 128, 512, 784 }, Wnum = 0, Onum = 0, Dnum = 0, dop;
	float* del, * delw, * weight, * Bweight, * out, * Inp, * Oout;
	clock_t t1;
	std::string filename;

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum << std::endl;
	std::cout << std::endl;

	float* weights = new float[NeuralSum];
	float* InputDataArr = new float[n[0]];
	float* outO = new float[n[layer - 1]];

	for (int i = 0; i < n[layer - 1]; i++)
		outO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * sizeof(float));

	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\0\\1 (1).png");
	cv::Mat result(image.rows, image.cols, CV_8UC1);
	Input(n, layer, outO, Oout, image);

	DataCheck(WeightSum, weight, delw);

	t1 = clock();
	for (int adm = 0; adm < 100; adm++) {
		std::cout << "Iter #" << adm + 1 << std::endl;
		for (int l = 0; l < 5000; l++) {
			for (int num = 0; num < 1; num++) {
				for (int k = 0; k < 2; k++) {
					cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num + 1) + ").png");
					/*cv::imshow("Out1", image);
					cv::waitKey(1);*/
					InputDataArr[k] = 1;
					hipMemcpy(Inp, InputData, n[0] * sizeof(float), hipMemcpyHostToDevice);
					InputData << <n[0], 1 >> > (Inp, out, n[0]);
					
					Input(n, layer, outO, Oout, image);
					Iteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
					//Out(NeuralSum, layer, n, weights, out, result);
					InputDataArr[k] = 0;
				}
			}
		}
		float* Bweight = new float[WeightSum];
		float* Bdelw = new float[WeightSum];
		hipMemcpy(Bweight, weight, WeightSum * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(Bdelw, delw, WeightSum * sizeof(float), hipMemcpyDeviceToHost);
		std::ofstream fweight("E:\\Foton\\ngnl_data\\backup\\weight.dat");
		std::ofstream fdelw("E:\\Foton\\ngnl_data\\backup\\delw.dat");

		for (int i = 0; i < WeightSum; i++) {
			fweight << Bweight[i] << " ";
			fdelw << Bdelw[i] << " ";
			//fout << i << " ";
		}
		std::cout << "Backup" << std::endl;
		delete[] Bweight;
		delete[] Bdelw;

		std::ofstream config("E:\\Foton\\ngnl_data\\backup\\config.txt");
		config << 1;
		config.close();
	}
}
