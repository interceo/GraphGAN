#include "hip/hip_runtime.h"
#include <windows.h>
#include <iostream>
#include <cmath>
#include "func.cuh"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <ctime>
#include <fstream>
#include <cstring>

using namespace std;
using namespace cv;

const int coat = 4;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[coat] = { 2, 3, 5, 1 }, Wnum = 0, Onum = 0, Dnum = 0;
	float* del, * delw = NULL, * weight, * out, * Inp, * Oout;
	int addvar, dop = 0;
	float pixel = 0;
	clock_t t1, t2;
	string name = "E:\\Foton\\ngnl_data\\training\\", filename;

	for (int i = 0; i < coat; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum << std::endl;

	for (int i = 0; i < coat - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum << std::endl;
	
	for (int i = 0; i < coat - 1; i++)
		dop = n[i] + dop;

	float* weights = new float[WeightSum];
	float* InputDataArr = new float[n[0]];
	float* outO = new float[n[coat - 1]];
	/*for (int i = 0; i < n[0]; i++)
		InputDataArr[i] = 0;
	InputDataArr[0] = 0.7;*/
	//InputDataArr[1] = 0.3;
	/*InputDataArr[2] = 1;
	InputDataArr[3] = 0.563;*/

	/*for (int i = 0; i < n[coat - 1]; i++)
		outO[i] = 0;
	outO[0] = 0;*/
	//outO[1] = 1;

	hipMalloc((void**)&out, NeuralSum * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * sizeof(float));
	hipMalloc((void**)&Oout, n[coat - 1] * sizeof(float));

	hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Oout, outO, n[coat - 1] * sizeof(float), hipMemcpyHostToDevice);

	WeightCreation << <WeightSum, 1 >> > (weight, WeightSum);
	//InputData << <n[0], 1 >> > (Inp, out, n[0]);

	t1 = clock();
	for (int ad = 0; ad < 5000; ad++) {
		for (int num = 0; num < n[coat - 2]; num++) {
			ifstream nam(name + to_string(num) + ".txt");
			for (int k = 0; k < 2; k++) {
				/*nam >> filename;
				outO[num] = 1;
				Mat image = imread(name + to_string(num) + "\\" + filename);
				for (int i = 0; i < image.cols; i++) {
					for (int j = 0; j < image.rows; j++) {
						for (int p = 0; p < 3; p++) {
							pixel = pixel + image.at<Vec3b>(i, j)[p];
						}
						InputDataArr[i * image.rows + j] = pixel / 765;
						pixel = 0;
					}
				}*/

				/*hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(Oout, outO, n[coat - 1] * sizeof(float), hipMemcpyHostToDevice);*/
				//InputData << <n[0], 1 >> > (Inp, out, n[0]);

				InputDataArr[0] = k;
				InputDataArr[1] = 1 - k;
				hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
				InputData << <n[0], 1 >> > (Inp, out, n[0]);
				outO[0] = 1 - k;
				hipMemcpy(Oout, outO, n[coat - 1] * sizeof(float), hipMemcpyHostToDevice);

				for (int i = 0; i < coat - 1; i++) {
					Sumfunc << <n[i + 1], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);										//int coat, int Wnum, int Onum, float* weight, float* out
					Wnum = Wnum + n[i] * n[i + 1];
					Onum = Onum + n[i];
				}

				Onum = NeuralSum - n[coat - 1];
				Delta << <n[coat - 1], 100 >> > (Oout, out, del, Onum, n[coat - 1]);
				Wnum = WeightSum;

				for (int j = 0; j < coat - 2; j++) {
					//addvar = coat - 1 - j;
					Onum = Onum - n[coat - 2 - j];
					Wnum = Wnum - n[coat - 2 - j] * n[coat - 1 - j];
					DeltaN << <n[coat - 2 - j], 1 >> > (Dnum, Wnum, Onum, del, weight, out, n[coat - 1 - j], n[coat - 2 - j]);					    //int Dnum, int Wnum, int Onum, float* del, float* weight, float* out
					Dnum = Dnum + n[coat - 1 - j];
				}

				Wnum = WeightSum;
				Dnum = 0;
				Onum = NeuralSum - n[coat - 1];

				for (int j = 0; j < coat - 1; j++) {
					Onum = Onum - n[coat - 2 - j];
					//addvar = coat - 1 - j;
					Wnum = Wnum - n[coat - 1 - j] * n[coat - 2 - j];
					Deltaw << < n[coat - 2 - j], 1 >> > (weight, del, out, delw, Dnum, Onum, Wnum, n[coat - 1 - j], n[coat - 2 - j]);						//float* weight, float* del, float* out, float* delw, int Dnum, int Onum, int Wnum, int coat
					//std::cout << Wnum << endl;
					Dnum = Dnum + n[coat - 1 - j];
				}

				Wnum = 0;
				Onum = 0;
				Dnum = 0;
				//outO[num] = 0;

				hipMemcpy(weights, out, (NeuralSum) * sizeof(float), hipMemcpyDeviceToHost);
				for (int i = dop; i < NeuralSum; i++) cout << weights[i] << endl;
			}
			cout << endl;
		}
	}

	/*for (int num = 0; num < n[coat - 1]; num++) {
		Wnum = 0;
		Onum = 0;
		ifstream nam(name + to_string(num) + ".txt");
		nam >> filename;
		//std::cout << filename << endl;
		Mat image = imread(name + to_string(num) + "\\" + filename);
		/*cout << name + to_string(num) + "\\" + filename << endl;
		namedWindow("Display");
		imshow("Display", image);
		waitKey(1);
		Sleep(100);*/
		/*for (int i = 0; i < image.cols; i++) {
			for (int j = 0; j < image.rows; j++) {
				for (int p = 0; p < 3; p++) {
					pixel = pixel + image.at<Vec3b>(i, j)[p];
				}
				InputDataArr[i * image.rows + j] = pixel / 765;
				pixel = 0;
				//cout << InputDataArr[i * image.rows + j] << endl;
			}
		}

		hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
		//hipMemcpy(Oout, outO, n[coat - 1] * sizeof(float), hipMemcpyHostToDevice);
		InputData << <n[0], 1 >> > (Inp, out, n[0]);

		for (int i = 0; i < coat - 1; i++) {
			Sumfunc << <n[i + 1], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);										//int coat, int Wnum, int Onum, float* weight, float* out
			Wnum = Wnum + n[i] * n[i + 1];
			Onum = Onum + n[i];
		}

		hipMemcpy(weights, out, (NeuralSum) * sizeof(float), hipMemcpyDeviceToHost);
		for (int i = dop; i < NeuralSum; i++) cout << weights[i] << endl;
		cout << endl;
	}*/
	std::cout << "Time: " << clock() - t1 << std::endl;
	/*hipMemcpy(weights, out, (NeuralSum) * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = dop; i < NeuralSum; i++) cout << weights[i] << endl;*/
	//Sleep(5000);
}
