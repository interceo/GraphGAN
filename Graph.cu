#include <windows.h>
#include <iostream>
#include <cmath>
#include "func.cuh"

using namespace std;

const int coat = 4;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[coat] = { 3, 4, 2, 3 }, Wnum = 0, Onum = 0;
	float* del = NULL, * delw = NULL, * weight, * out, * Inp, * Oout;
	int addvar;

	for (int i = 0; i < coat; i++)   
		NeuralSum = n[i] + NeuralSum;

	for (int i = 0; i < coat - 1; i++) 
		WeightSum = n[i] * n[i + 1] + WeightSum;

	float* weights = new float[NeuralSum];
	float* InputDataArr = new float[n[0]];
	float* outO = new float[n[coat - 1]];
	InputDataArr[0] = 0.7;
	InputDataArr[1] = 0.3;
	InputDataArr[2] = 1;

	outO[0] = 0.4;
	outO[1] = 1;
	outO[2] = 0.6;
	addvar = n[coat - 1];
	/*outO[3] = 0.785;
	outO[4] = 1;*/

	hipMalloc((void**)&out, NeuralSum * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * sizeof(float));
	hipMalloc((void**)&Oout, addvar * sizeof(float));

	hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(Oout, outO, n[coat - 1] * sizeof(float), hipMemcpyHostToDevice);

	WeightCreation << <WeightSum, 1 >> > (weight, WeightSum);
	InputData << <n[0], 1 >> > (Inp, out, n[0]);
 
	hipMemcpy(weights, out, NeuralSum * sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(out, InputData, NeuralS * sizeof(float), hipMemcpyHostToDevice);
	for (int i = 0; i < n[0]; i++) cout << weights[i] << endl;

	cout << "Num #1" << endl;

	for (int i = 0; i < coat - 1; i++) {
		Sumfunc << <n[i], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);										//int coat, int Wnum, int Onum, float* weight, float* out
		Wnum = Wnum + n[i] * n[i + 1];
		Onum = Onum + n[i];
	} 
	hipMemcpy(weights, out, NeuralSum * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < n[coat - 1]; i++) cout << weights[Onum + i] << endl;

	addvar = n[coat - 1];
	Delta << <n[addvar], 1 >> > (weight, Oout, out, del, Onum, n[addvar]);									    //float* weight, float* outO, float* out, float* del, int Onum, int size

	/*hipMemcpy(weights, del, n[coat - 1] * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < n[coat - 1]; i++) cout << weights[i] << endl;*/

	for (int i = coat - 1; i > 0; i--) {
		for (int j = 0; j < n[i]; j++) {
			Deltaw << <n[coat - i], 1 >> > (n[coat - i], delw, n[coat - i], del, out, Wnum, Onum, weight);			//int size, float* delw, int coat, float* del, float* out, int Wnum, int Onum, float* weight
			Wnum = Wnum + n[i];
		}
		Wnum = Wnum - n[i] * n[i - 1];
		Onum = Onum - n[i - 1];
	}
  }
