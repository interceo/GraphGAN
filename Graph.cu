#include "hip/hip_runtime.h"
#include <windows.h>
#include <iostream>
#include <cmath>
#include "func.cuh"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <ctime>
#include <fstream>
#include <cstring>

using namespace cv;

const int layer = 2;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 1, 784 }, Wnum = 0, Onum = 0, Dnum = 0, dop = 0;
	float* del, * delw, * weight, * out, * Inp, * Oout, pixel = 0;
	clock_t t1;
	
	Mat result(28, 28, CV_8UC1);

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum << std::endl;

	float* weights = new float[WeightSum];
	float* InputDataArr = new float[n[0]];
	float* outO = new float[n[layer - 1]];

	for (int i = 0; i < n[layer - 1]; i++)
		outO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * sizeof(float));

	WeightGen << <WeightSum, 1 >> > (weight, WeightSum);
	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	InputDataArr[0] = 0.524;
	hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
	InputData << <n[0], 1 >> > (Inp, out, n[0]);

	t1 = clock();
	for (int ad = 0; ad < 1; ad++) {
		for (int num = 0; num < 500; num++) {
			for (int k = 0; k < 2; k++) {
				Mat image = imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k * 6) + "\\" + std::to_string(1 + k * 12) + ".png");
				for (int i = 0; i < 28; i++) {
					for (int j = 0; j < 28; j++) {
						float per = 0;
						per = image.at<Vec3b>(i, j)[0];
						per = per / 255;
						outO[i * 28 + j] = per;
					}
				}
				hipMemcpy(Oout, outO, n[layer - 1] * sizeof(float), hipMemcpyHostToDevice);
				
				//Clayer << < 49, 1 >> > (weight, out, n[0]);
				for (int i = 0; i < (layer - 1); i++) {
					Sumfunc << <n[i + 1], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);										//int layer, int Wnum, int Onum, float* weight, float* out
					Wnum = Wnum + n[i] * n[i + 1];
					Onum = Onum + n[i];
				}

				Onum = NeuralSum - n[layer - 1];
				Delta << <n[layer - 1], 1 >> > (Oout, out, del, Onum, n[layer - 1]);
				Wnum = WeightSum;

				for (int j = 0; j < layer - 1; j++) {
					Onum = Onum - n[layer - 2 - j];
					Wnum = Wnum - n[layer - 2 - j] * n[layer - 1 - j];
					DeltaN << <n[layer - 2 - j], 1 >> > (Dnum, Wnum, Onum, del, weight, out, n[layer - 1 - j], n[layer - 2 - j]);					    //int Dnum, int Wnum, int Onum, float* del, float* weight, float* out
					Dnum = Dnum + n[layer - 1 - j];
				}

				Wnum = WeightSum;
				Dnum = 0;
				Onum = NeuralSum - n[layer - 1];

				for (int j = 0; j < layer - 1; j++) {
					Onum = Onum - n[layer - 2 - j];
					Wnum = Wnum - n[layer - 1 - j] * n[layer - 2 - j];
					Deltaw << < n[layer - 2 - j], 1 >> > (weight, del, out, delw, Dnum, Onum, Wnum, n[layer - 1 - j], n[layer - 2 - j]);				//float* weight, float* del, float* out, float* delw, int Dnum, int Onum, int Wnum, int layer, int n
					Dnum = Dnum + n[layer - 1 - j];
				}

				//Dnum = NeuralSum - n[0] - n[1];
				//ConvDeltaW << < 49, 1 >> > (weight, out, del, delw, Dnum);

				Wnum = 0;
				Dnum = 0;
				Onum = 0;
				Onum = NeuralSum - n[layer - 1];
				hipMemcpy(weights, out, NeuralSum * sizeof(float), hipMemcpyDeviceToHost);
				for (int i = 0; i < 28; i++) {
					for (int j = 0; j < 28; j++) {
						float per = 0;
						per = weights[Onum + i * 28 + j];
						per = ceil(per * 255);
						//std::cout << per << std::endl;
						result.at<uchar>(i, j) = per;
					}
				}
				imshow("admin", result);
				waitKey(1);
				Onum = 0;
			}
		}

	}

	for (int i = 0; i < 100; i++) {
		std::cin >> InputDataArr[0];
		hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
		InputData << <n[0], 1 >> > (Inp, out, n[0]);
		for (int i = 0; i < (layer - 1); i++) {
			Sumfunc << <n[i + 1], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);										//int layer, int Wnum, int Onum, float* weight, float* out
			Wnum = Wnum + n[i] * n[i + 1];
			Onum = Onum + n[i];
		}

		Onum = NeuralSum - n[layer - 1];
		hipMemcpy(weights, out, NeuralSum * sizeof(float), hipMemcpyDeviceToHost);
		for (int i = 0; i < 28; i++) {
			for (int j = 0; j < 28; j++) {
				float per = 0;
				per = weights[Onum + i * 28 + j];
				per = ceil(per * 255);
				result.at<uchar>(i, j) = per;
			}
		}
		imshow("admin", result);
		waitKey(1);
		Onum = 0;
		Wnum = 0;
	}
	std::cout << "Time: " << clock() - t1 << std::endl;
}
