#include "hip/hip_runtime.h"
#include <windows.h>
#include <iostream>
#include <cmath>
#include "func.cuh"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <ctime>
#include <fstream>
#include <cstring>

using namespace cv;

const int layer = 3;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 2, 2, 1 }, Wnum = 0, Onum = 0, Dnum = 0, dop = 0;
	float* del, * delw, * weight, * out, * Inp, * Oout, pixel = 0;
	clock_t t1;
	//std::string name = "E:\\Foton\\ngnl_data\\training\\", filename;

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum << std::endl;

	float* weights = new float[WeightSum];
	float* InputDataArr = new float[n[0]];
	float* outO = new float[n[layer - 1]];

	for (int i = 0; i < n[layer - 1]; i++)
		outO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * sizeof(float));

	WeightGen << <WeightSum, 1 >> > (weight, WeightSum);
	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	t1 = clock();
	for (int ad = 0; ad < 1; ad++) {
		for (int num = 0; num < 500; num++) {
			//std::ifstream nam(name + std::to_string(num) + ".txt");
			for (int k = 0; k < 2; k++) {
				InputDataArr[0] = 1 - k;
				InputDataArr[1] = k;
				hipMemcpy(Inp, InputDataArr, n[0] * sizeof(float), hipMemcpyHostToDevice);
				InputData << <n[0], 1 >> > (Inp, out, n[0]);
				outO[0] = 1 - k;
				hipMemcpy(Oout, outO, n[layer - 1] * sizeof(float), hipMemcpyHostToDevice);

				//Clayer << < 49, 1 >> > (weight, out, n[0]);
				for (int i = 0; i < (layer - 1); i++) {
					Sumfunc << <n[i + 1], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);										//int layer, int Wnum, int Onum, float* weight, float* out
					Wnum = Wnum + n[i] * n[i + 1];
					Onum = Onum + n[i];
				}

				Onum = NeuralSum - n[layer - 1];
				Delta << <n[layer - 1], 1 >> > (Oout, out, del, Onum, n[layer - 1]);
				Wnum = WeightSum;

				for (int j = 0; j < layer - 1; j++) {
					Onum = Onum - n[layer - 2 - j];
					Wnum = Wnum - n[layer - 2 - j] * n[layer - 1 - j];
					DeltaN << <n[layer - 2 - j], 1 >> > (Dnum, Wnum, Onum, del, weight, out, n[layer - 1 - j], n[layer - 2 - j]);					    //int Dnum, int Wnum, int Onum, float* del, float* weight, float* out
					Dnum = Dnum + n[layer - 1 - j];
				}

				Wnum = WeightSum;
				Dnum = 0;
				Onum = NeuralSum - n[layer - 1];

				for (int j = 0; j < layer - 1; j++) {
					Onum = Onum - n[layer - 2 - j];
					Wnum = Wnum - n[layer - 1 - j] * n[layer - 2 - j];
					Deltaw << < n[layer - 2 - j], 1 >> > (weight, del, out, delw, Dnum, Onum, Wnum, n[layer - 1 - j], n[layer - 2 - j]);				//float* weight, float* del, float* out, float* delw, int Dnum, int Onum, int Wnum, int layer, int n
					Dnum = Dnum + n[layer - 1 - j];
				}

				/*Dnum = NeuralSum - n[0] - n[1];
				ConvDeltaW << < n[1] / 16, 1 >> > (weight, out, del, delw, Dnum, n[1] / 49);*/

				Wnum = 0;
				Onum = 0;
				Dnum = 0;

				hipMemcpy(weights, out, (NeuralSum) * sizeof(float), hipMemcpyDeviceToHost);
				for (int i = (NeuralSum - n[layer - 1]); i < NeuralSum; i++) {
					//std::cout << InputDataArr[0] << "" << InputDataArr[1] << " ";
					std::cout << weights[i] << " " << outO[i] << std::endl;;
				}
			}
			std::cout << std::endl;
		}

	}
	std::cout << "Time: " << clock() - t1 << std::endl;
}
