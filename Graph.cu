#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <fstream>
#include <ctime>

#include "func.cuh"

const int layer = 3;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 32, 256, 784 };
	float* del, * delw, * weight, * out, * Inp, * Oout;
	clock_t t1;
	std::string filename;

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum * 3 << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum * 3 << std::endl;
	std::cout << std::endl;

	//float* InputDataArr = new float[n[0]];
	float* outO = new float[n[layer - 1] * 3];

	for (int i = 0; i < n[layer - 1]; i++)
		outO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * 3 * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * 3 * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * 3 * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * 3 * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * 3 * sizeof(float));

	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\0\\1 (1).png");
	/*cv::imshow("Out", image);
	cv::waitKey(10000);*/
	cv::Mat result(image.rows, image.cols, CV_8UC3);
	Input(n[layer - 1], outO, Oout, image);
	DataCheck(WeightSum, weight, delw);

	t1 = clock();
	for (int adm = 0; adm < 1000; adm++) {
		std::cout << "Iter #" << adm + 1 << std::endl;
		for (int l = 0; l < 1; l++) {
			for (int num = 0; num < 5000; num++) {
				for (int k = 0; k < 10; k++) {
					cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num + 1) + ").png");
					InputGen(n[0], NeuralSum, Inp, out);
					Iteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
					Out(NeuralSum, layer, n, out, result);
				}
			}
		}
		float* Bweight = new float[WeightSum];
		float* Bdelw = new float[WeightSum];
		hipMemcpy(Bweight, weight, WeightSum * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(Bdelw, delw, WeightSum * sizeof(float), hipMemcpyDeviceToHost);
		std::ofstream fweight("E:\\Foton\\ngnl_data\\backup\\weight.dat");
		std::ofstream fdelw("E:\\Foton\\ngnl_data\\backup\\delw.dat");

		for (int i = 0; i < WeightSum; i++) {
			fweight << Bweight[i] << " ";
			fdelw << Bdelw[i] << " ";
		}
		std::cout << "Backup" << std::endl;
		delete[] Bweight;
		delete[] Bdelw;
		fweight.close();
		fdelw.close();

		std::ofstream config("E:\\Foton\\ngnl_data\\backup\\config.txt");
		config << 1;
		config.close();
	}
}
