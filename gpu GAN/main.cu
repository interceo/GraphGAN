#include "hip/hip_runtime.h"
//Create by Stasan

#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <fstream>
#include <ctime>

#include "func.cuh"

const int layer = 4, RGB = 1;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 4, 16, 4, 784 }, int dop = 0,
		DisWeightSum = 0, DisNeuralSum = 0, nc[layer] = { 784, 64, 16, 1 };
	float * del, * delw, * weight, * out, * Inp, * Oout, * DOout,
		  * Disdel, * Disdelw, * Disweight, * Disout, * DisInp, * DisOout;
	clock_t t1;
	std::string filename;

	/*---------перевести в const---------*/
	/*определение основных переменных Gen*/

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum * RGB << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum * RGB << std::endl;

	for (int i = 0; i < layer; i++)
		DisNeuralSum = nc[i] + DisNeuralSum;
	std::cout << "Neurals: " << DisNeuralSum * RGB << std::endl;

	for (int i = 0; i < layer - 1; i++)
		DisWeightSum = nc[i] * nc[i + 1] + DisWeightSum;
	std::cout << "Weights: " << DisWeightSum * RGB << std::endl;
	std::cout << std::endl;

	/*---------перевести в const---------*/

	float* outO = new float[n[layer - 1] * RGB];                //оптимальный вариант генератора в RAM
	float* DisoutO = new float[nc[layer - 1] * RGB];            //оптимальный вариант дискриминатора в RAM
	float* DisResult = new float[nc[layer - 1] * RGB];          //решение дискриминатора

	for (int i = 0; i < n[layer - 1] * RGB; i++)
		outO[i] = 0;
	for (int i = 0; i < nc[layer - 1] * RGB; i++)
		DisoutO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * RGB * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * RGB * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * RGB * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * RGB * sizeof(float));                              //входные данные генератора в GPU RAM
	hipMalloc((void**)&Oout, n[layer - 1] * RGB * sizeof(float));                     //оптимальный вариант генератора в GPU RAM
	hipMalloc((void**)&DOout, RGB * sizeof(float));                                   //оптимальный вариант генератора в GPU RAM

	hipMalloc((void**)&Disout, DisNeuralSum * RGB * sizeof(float));
	hipMalloc((void**)&Disdel, (DisNeuralSum - nc[0]) * sizeof(float));
	hipMalloc((void**)&Disweight, DisWeightSum * RGB * sizeof(float));
	hipMalloc((void**)&Disdelw, DisWeightSum * RGB * sizeof(float));
	hipMalloc((void**)&DisInp, nc[0] * RGB * sizeof(float));                          //входные данные дискриминатора в GPU RAM
	hipMalloc((void**)&DisOout, nc[layer - 1] * RGB * sizeof(float));                 //оптимальный вариант дискриминатора в GPU RAM

	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);
	DelwNull << < DisWeightSum, 1 >> > (Disdelw, DisWeightSum);

	cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\0\\1 (9).png");
	cv::Mat result(image.rows, image.cols, CV_8UC1);

	//InputOutImage(n[layer - 1], outO, Oout, image, RGB);
	DataCheck(WeightSum, weight, delw, 0, RGB);
	DataCheck(DisWeightSum, Disweight, Disdelw, 1, RGB);

	t1 = clock();
	for (int adm = 0; adm < 1000; adm++) {
		std::cout << "Iter #" << adm + 1 << std::endl;
		for (int l = 0; l < 1; l++) {
			for (int num = 0; num < 1; num++) {
				for (int k = 0; k < 10; k++) {
					//---------Input image data---//
					//---------SumFunc------------//
					//---------Out data-----------//
					//---------Opt out data-------//
					//---------Iter---------------//

					//---------Input result data--//
					//---------SumFunc------------//
					//---------Out data-----------//
					//---------Opt out data-------//
					//---------Iter---------------//

					cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(1) + "\\1 (" + std::to_string(k + 1) + ").png");
					InputInputImage(nc[0], Disout, DisoutO, DisInp, image, DisNeuralSum, RGB);
					GlobalSumFunc(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, RGB);
					//ImageResult(DisNeuralSum, Disout, nc[layer - 1], RGB);
					OptDis(DisoutO, nc[layer - 1], RGB, DisOout, 1);
					DisIteration(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, Disdelw, DisOout, DisoutO, Disdel, RGB);

					//Random(nc[0], DisNeuralSum, DisInp, Disout, RGB);
					Random(n[0], NeuralSum, Inp, out, RGB);
					GlobalSumFunc(n, layer, NeuralSum, WeightSum, weight, out, RGB);
					Convert(DisInp, nc[0], NeuralSum, DisNeuralSum, out, RGB);
					for (int i = 0; i < RGB; i++)
						InputData << <nc[0], 1 >> > (DisInp, Disout, nc[0], i, DisNeuralSum);

					GlobalSumFunc(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, RGB);
					//ImageResult(DisNeuralSum, Disout, nc[layer - 1], RGB);
					OptDis(DisoutO, nc[layer - 1], RGB, DisOout, 0);
					DisIteration(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, Disdelw, DisOout, DisoutO, Disdel, RGB);
				}
				std::cout << "data";
				for (int k = 0; k < 10000; k++) {
					Random(n[0], NeuralSum, Inp, out, RGB);
					GlobalSumFunc(n, layer, NeuralSum, WeightSum, weight, out, RGB);
					Convert(DOout, nc[layer - 1], DisNeuralSum, 0, Disout, RGB);
					GenIteration(n, layer, NeuralSum, WeightSum, weight, out, delw, DOout, outO, del, RGB);
					Out(NeuralSum, layer, n, weight, out, result);
				}
			}
		}
		Backup(WeightSum, weight, delw, 0, RGB);
		Backup(DisWeightSum, Disweight, Disdelw, 1, RGB);
	}
}
