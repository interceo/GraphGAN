#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "hip/hip_runtime.h"
#include ""

using namespace std;
using namespace cv;

__global__ void Inversion_CUDA(unsigned char* Image, int Channels) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = 255 - Image[idx + i];
	}
}

int main() {
	Mat Input_Image = imread("E:\\Foton\\ngnl_data\\training\\0\\1.png");

	cout << "Height: " << Input_Image.rows << ", Width: " << Input_Image.cols << ", Channels: " << Input_Image.channels() << endl;

	unsigned char* Dev_Input_Image = NULL;

	hipMalloc((void**)&Dev_Input_Image, Input_Image.rows * Input_Image.cols * Input_Image.channels());
	hipMemcpy(Dev_Input_Image, Input_Image.data, Input_Image.rows * Input_Image.cols * Input_Image.channels(), hipMemcpyHostToDevice);

	dim3 Grid_Image(Input_Image.rows * Input_Image.cols);
	Inversion_CUDA << <Grid_Image, 1 >> > (Dev_Input_Image, Input_Image.channels());
	hipMemcpy(Input_Image.data, Dev_Input_Image, Input_Image.rows * Input_Image.cols * Input_Image.channels(), hipMemcpyDeviceToHost);

	hipFree(Dev_Input_Image);

	imwrite("E:\\image.png", Input_Image);
	return 0;
}
