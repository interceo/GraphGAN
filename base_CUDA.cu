#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "hip/hip_runtime.h"
#include ""

using namespace std;
using namespace cv;

__global__ void Inversion_CUDA(unsigned char* Image, int Channels) {
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = 255 - Image[idx + i];
	}
}

void Image_Inversion_CUDA(unsigned char* Input_Image, int Height, int Width, int Channels) {
	unsigned char* Dev_Input_Image = NULL;

	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image(Width, Height);
	Inversion_CUDA << <Grid_Image, 1 >> > (Dev_Input_Image, Channels);
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	hipFree(Dev_Input_Image);
}

int main() {
	Mat Input_Image = imread("image.png");

	cout << "Height: " << Input_Image.rows << ", Width: " << Input_Image.cols << ", Channels: " << Input_Image.channels() << endl;
	system("pause");

	Image_Inversion_CUDA(Input_Image.data, Input_Image.rows, Input_Image.cols, Input_Image.channels());

	imwrite("image.png", Input_Image);
	return 0;
}
