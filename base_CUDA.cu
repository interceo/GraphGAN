#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "hip/hip_runtime.h"
#include ""

using namespace std;
using namespace cv;

__global__ void Inversion_CUDA(unsigned char* Image, int Channels, int* admin) {
	int pixel = 0;
	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) pixel = pixel + Image[idx + i];
	admin[x + y * gridDim.x] = pixel;
}

int main() {
	//Mat Input_Image = imread("E:\\Foton\\ngnl_data\\training\\0\\1.png");
	Mat Input_Image = imread("E:\\photo.png");
	unsigned int N = Input_Image.rows * Input_Image.cols;
	unsigned int K = N * 3;

	cout << "Height: " << N << ", Width: " << Input_Image.cols << ", Channels: " << Input_Image.channels() << endl;

	unsigned char* Dev_Input_Image = NULL;
	int l = 0;
	int pixels = 0;
	int* pixel;
	int* Image = new int[N];

	hipMalloc((void**)&Dev_Input_Image, K);
	hipMemcpy(Dev_Input_Image, Input_Image.data, K, hipMemcpyHostToDevice);

	hipMalloc((void**)&pixel, N);

	dim3 Grid_Image(Input_Image.rows * Input_Image.cols);
	Inversion_CUDA << <Grid_Image, 1 >> > (Dev_Input_Image, Input_Image.channels(), pixel);
	/*for (int i = 0; i < Input_Image.rows; i++) {
		for (int k = 0; k < Input_Image.cols; k++) {
			for (int j = 0; j < 3; j++) pixels = pixels + Input_Image.at<Vec3b>(i, k)[j];
			Image[l] = pixels;
			l++;
			pixels = 0;
		}
	}*/

	//hipMemcpy(Input_Image.data, Dev_Input_Image, Input_Image.rows * Input_Image.cols * Input_Image.channels(), hipMemcpyDeviceToHost);
	hipMemcpy(Image, pixel, N, hipMemcpyDeviceToHost);

	hipFree(Dev_Input_Image);
	hipFree(pixel);

	//for (int i = 0; i < N; i++) cout << Image[i] << endl;

	//imwrite("E:\\image.png", Input_Image);
	return 0;
}
