#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <fstream>
#include <ctime>

#include "func.cuh"

const int layer = 3;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 16, 64, 784 }, int dop = 0, RGB = 3,
		DisWeightSum = 0, DisNeuralSum = 0, nc[layer] = { 784, 64, 1 };
	float * del, * delw, * weight, * out, * Inp, * Oout,
		  * Disdel, * Disdelw, * Disweight, * Disout, * DisInp, * DisOout;
	clock_t t1;
	std::string filename;

	/*---------перевести в const---------*/
	/*определение основных переменных Gen*/

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum * RGB << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum * RGB << std::endl;

	for (int i = 0; i < layer; i++)
		DisNeuralSum = nc[i] + DisNeuralSum;
	std::cout << "Neurals: " << DisNeuralSum * RGB << std::endl;

	for (int i = 0; i < layer - 1; i++)
		DisWeightSum = nc[i] * nc[i + 1] + DisWeightSum;
	std::cout << "Weights: " << DisWeightSum * RGB << std::endl;
	std::cout << std::endl;

	/*---------перевести в const---------*/

	float* outO = new float[n[layer - 1] * RGB];
	float* DisoutO = new float[nc[layer - 1] * RGB];

	for (int i = 0; i < n[layer - 1] * RGB; i++)
		outO[i] = 0;
	for (int i = 0; i < nc[layer - 1] * RGB; i++)
		DisoutO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * RGB * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * RGB * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * RGB * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * RGB * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * 1 * sizeof(float));

	hipMalloc((void**)&Disout, DisNeuralSum * RGB * sizeof(float));
	hipMalloc((void**)&Disdel, (DisNeuralSum - nc[0]) * sizeof(float));
	hipMalloc((void**)&Disweight, DisWeightSum * RGB * sizeof(float));
	hipMalloc((void**)&Disdelw, DisWeightSum * RGB * sizeof(float));
	hipMalloc((void**)&DisInp, nc[0] * RGB * sizeof(float));
	hipMalloc((void**)&DisOout, nc[layer - 1] * RGB * sizeof(float));

	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);
	DelwNull << < DisWeightSum, 1 >> > (Disdelw, DisWeightSum);

	cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\0\\1 (9).png");
	cv::Mat result(image.rows, image.cols, CV_8UC1);
	InputOutImage(n[layer - 1], outO, Oout, image);
	DataCheck(WeightSum, weight, delw, 0);
	DataCheck(DisWeightSum, Disweight, Disdelw, 1);

	t1 = clock();
	for (int adm = 0; adm < 1000; adm++) {
		std::cout << "Iter #" << adm + 1 << std::endl;
		for (int l = 0; l < 1; l++) {
			for (int num = 0; num < 1; num++) {
				/*for (int k = 0; k < 5000; k++) {
					/*---------Gen from Iter---------*/
					/*InputDiffer(dop, n, layer, WeightSum, nc[0], DisNeuralSum, NeuralSum, DisInp, Disout, weight, out, outO, dop, Inp);
					GlobalSumFunc(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout);
					for (int i = 0; i < RGB; i++)
						DisoutO[i] = 0;
					DisIteration(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, Disdelw, DisOout, DisoutO, Disdel, DisInp);
					/*ImageResult(DisNeuralSum, Disout, nc[layer - 1]);
					std::cout << std::endl;*/

					/*---------Opt from Iter---------*/
					/*cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(0) + "\\1 (" + std::to_string(k + 1) + ").png");
					InputInputImage(nc[0], Disout, outO, DisInp, image, DisNeuralSum);
					GlobalSumFunc(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout);
					for (int i = 0; i < RGB; i++)
						DisoutO[i] = 1;
					DisIteration(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, Disdelw, DisOout, DisoutO, Disdel, DisInp);
					/*ImageResult(DisNeuralSum, Disout, nc[layer - 1]);
					std::cout << std::endl;*/
				/*}
				dop = 1;*/

				for (int k = 0; k < 5000; k++) {
					/*cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num + 1) + ").png");
					Input(n[layer - 1], outO, Oout, image);*/
					// Random(n[0], NeuralSum, Inp, out);
					/*InputDiffer(dop, n, layer, WeightSum, nc[0], DisNeuralSum, NeuralSum, DisInp, Disout, weight, out, outO, dop, Inp);
					GlobalSumFunc(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout);
					ImageOpt(DisNeuralSum, Disout, nc[layer - 1], outO);*/
					//InputOutImage(n[layer - 1], outO, Oout, image);
					Random(n[0], NeuralSum, Inp, out);
					GenIteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
					OutOutImage(NeuralSum, layer, n, out, result);
					//std::cout << WeightSum << std::endl;
				}
			}
		}
		Backup(WeightSum, weight, delw, 0);
		Backup(DisWeightSum, Disweight, Disdelw, 1);
	}
}
