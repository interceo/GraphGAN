#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <ctime>

#include "func.cuh"

const int layer = 2;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 1, 609750 }, Wnum = 0, Onum = 0, Dnum = 0, dop = 1;
	float* del, * delw, * weight, * out, * Inp, * Oout;
	clock_t t1;
	std::string filename;

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum << std::endl;

	float* weights = new float[NeuralSum];
	//float* data = new float[WeightSum];
	float* InputDataArr = new float[n[0]];
	float* outO = new float[n[layer - 1]];

	for (int i = 0; i < n[layer - 1]; i++)
		outO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * sizeof(float));

	WeightGen << <WeightSum, 1 >> > (weight, WeightSum);
	//WeightsGen(data, weight, WeightSum);
	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	cv::Mat image = cv::imread("E:\\1.jpg");
	cv::Mat result(image.rows, image.cols, CV_8UC1);
	OutputData(n, layer, outO, Oout, image, result, InputDataArr, Inp, out);
	cv::imshow("Out1", image);
	cv::waitKey(1);
	//cv::Mat result(28, 28, CV_8UC1);
	t1 = clock();
	for (int adm = 0; adm < 20; adm++) {
		std::cout << "Iter #" << adm << std::endl;
		for (int num = 0; num < 5000; num++) {
			for (int k = 0; k < 10; k++) {
				//cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num + 1) + ").png");
				//cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\0\\1 (1).png");
				//std::cout << "E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num + 1) + ").png";
				//OutputData(n, layer, outO, Oout, image, result, InputDataArr, Inp, out);
				/*outO[k] = 1;
				hipMemcpy(Oout, outO, n[layer - 1] * sizeof(float), hipMemcpyHostToDevice);*/

				//NumberInp(out, image, InputDataArr, n, layer, Inp);
				Iteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
				//NumberOut(out, weights, n, layer, NeuralSum);
				Out(NeuralSum, layer, n, weights, out, image, result);
				outO[k] = 0;
			}
		}
	}

	std::cout << "Time " << clock() - t1 << std::endl;

	for (int i = 0; i < 100; i++) {
		int Wnum = 0, Onum = 0, Dnum = 0;
		//std::cin >> filename;
		cv::Mat image = cv::imread("E:\\2.png");
		NumberInp(out, image, InputDataArr, n, layer, Inp);
		cv::imshow("Out", image);
		//Iteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
		for (int i = 0; i < (layer - 1); i++) {
			Sumfunc << <n[i + 1], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);
			Wnum = Wnum + n[i] * n[i + 1];
			Onum = Onum + n[i];
		}

		NumberOut(out, weights, n, layer, NeuralSum);
	}
}
