#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <fstream>
#include <ctime>

#include "func.cuh"

const int layer = 4;

int main() {
	int WeightSum = 0, NeuralSum = 0, DisWeightSum = 0, DisNeuralSum = 0, n[layer] = { 16, 64, 128, 784 }, nc[layer] = {784, 512, 64, 1};
	float* del, * delw, * weight, * out, * Inp, * Oout, * Disweight, * Disout, * Disdelw, * DisOout, * Disdel, * DisInp, * Dis;
	clock_t t1;
	std::string filename;

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum * 3 << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum * 3 << std::endl;

	for (int i = 0; i < layer; i++)
		DisNeuralSum = nc[i] + DisNeuralSum;
	std::cout << "Neurals: " << DisNeuralSum * 3 << std::endl;

	for (int i = 0; i < layer - 1; i++)
		DisWeightSum = nc[i] * nc[i + 1] + DisWeightSum;
	std::cout << "Weights: " << DisWeightSum * 3 << std::endl;
	std::cout << std::endl;

	float* outO = new float[n[layer - 1] * 3];
	float* DisoutO = new float[nc[layer - 1] * 3];

	/*for (int i = 0; i < n[layer - 1] * 3; i++)
		outO[i] = 0;*/

	hipMalloc((void**)&out, NeuralSum * 3 * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * 3 * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * 3 * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * 3 * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * 3 * sizeof(float));

	hipMalloc((void**)&Disout, DisNeuralSum * 3 * sizeof(float));
	hipMalloc((void**)&Disdel, (DisNeuralSum - nc[0]) * sizeof(float));
	hipMalloc((void**)&Disweight, DisWeightSum * 3 * sizeof(float));
	hipMalloc((void**)&Disdelw, DisWeightSum * 3 * sizeof(float));
	hipMalloc((void**)&DisInp, nc[0] * 3 * sizeof(float));
	hipMalloc((void**)&DisOout, nc[layer - 1] * 3 * sizeof(float));

	hipMalloc((void**)&Dis, nc[layer - 1] * 3 * sizeof(float));

	DelwNull << < WeightSum, 1 >> > (delw, WeightSum, 0);
	DelwNull << < DisWeightSum, 1 >> > (Disdelw, DisWeightSum, 0);

	cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\0\\1 (9).png");
	/*cv::imshow("Out", image);
	cv::waitKey(10000);*/
	cv::Mat result(image.rows, image.cols, CV_8UC3);
	Input(n[layer - 1], outO, Oout, image);
	DataCheck(WeightSum, weight, delw, 0);
	DataCheck(DisWeightSum, Disweight, Disdelw, 1);

	DelwNull << < nc[layer - 1], 1 >> > (Dis, nc[layer - 1], 1);

	t1 = clock();
	for (int adm = 0; adm < 1000; adm++) {
		std::cout << "Iter #" << adm + 1 << std::endl;
		for (int l = 0; l < 1; l++) {
			for (int num = 0; num < 5000; num++) {
				for (int k = 0; k < 1; k++) {
					InputGen(n[0], NeuralSum, Inp, out);
					SumGen(WeightSum, NeuralSum, DisNeuralSum, n, weight, out, result, Inp, Disout, layer);
					SumDis(DisWeightSum, DisNeuralSum, NeuralSum, nc, Disweight, Disout, Dis, image, Inp, layer);
					//cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num + 1) + ").png");
					//Input(n[layer - 1], outO, Oout, image);
					IterationGen(n, layer, NeuralSum, WeightSum, weight, out, delw, Dis, del);
					Out(NeuralSum, layer, n, out, result);
				}
			}
		}
		Backup(WeightSum, weight, delw, 0);

		for (int l = 0; l < 1; l++) {
			for (int num = 0; num < 5000; num++) {
				InputGen(n[0], NeuralSum, Inp, out);
				SumGen(WeightSum, NeuralSum, DisNeuralSum, n, weight, out, result, Inp, Disout, layer);

				//InputGen(nc[0], DisNeuralSum, DisInp, Disout);
				for (int i = 0; i < 3; i++)
					DisoutO[i] = 0.3;
				hipMemcpy(DisOout, DisoutO, nc[layer - 1] * 3 * sizeof(float), hipMemcpyHostToDevice);
				IterationGen(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, Disdelw, DisOout, Disdel);
				/*ImageResult(DisNeuralSum, Disout, nc[layer - 1]);
				std::cout << std::endl;*/

				for (int i = 0; i < 3; i++)
					DisoutO[i] = 1;
				hipMemcpy(DisOout, DisoutO, nc[layer - 1] * 3 * sizeof(float), hipMemcpyHostToDevice);
				cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(0) + "\\1 (" + std::to_string(num + 1) + ").png");
				/*cv::imshow("Out", image);
				cv::waitKey(1);*/
				InputImage(nc[0], Disout, DisInp, image, DisNeuralSum);
				IterationGen(nc, layer, DisNeuralSum, DisWeightSum, Disweight, Disout, Disdelw, DisOout, Disdel);
				/*ImageResult(DisNeuralSum, Disout, nc[layer - 1]);
				std::cout << std::endl;*/
			}
		}
		Backup(DisWeightSum, Disweight, Disdelw, 1);
	}
}
