#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <iostream>
#include <ctime>

#include "func.cuh"

const int layer = 3;

int main() {
	int WeightSum = 0, NeuralSum = 0, n[layer] = { 1, 20, 784 }, Wnum = 0, Onum = 0, Dnum = 0, dop = 1;
	float* del, * delw, * weight, * out, * Inp, * Oout, max = 10, min = -10;
	clock_t t1;
	std::string filename;

	for (int i = 0; i < layer; i++)
		NeuralSum = n[i] + NeuralSum;
	std::cout << "Neurals: " << NeuralSum << std::endl;

	for (int i = 0; i < layer - 1; i++)
		WeightSum = n[i] * n[i + 1] + WeightSum;
	std::cout << "Weights: " << WeightSum << std::endl;

	float* weights = new float[NeuralSum];
	float* InputDataArr = new float[n[0]];
	float* outO = new float[n[layer - 1]];

	for (int i = 0; i < n[layer - 1]; i++)
		outO[i] = 0;

	hipMalloc((void**)&out, NeuralSum * sizeof(float));
	hipMalloc((void**)&del, (NeuralSum - n[0]) * sizeof(float));
	hipMalloc((void**)&weight, WeightSum * sizeof(float));
	hipMalloc((void**)&delw, WeightSum * sizeof(float));
	hipMalloc((void**)&Inp, n[0] * sizeof(float));
	hipMalloc((void**)&Oout, n[layer - 1] * sizeof(float));

	WeightGen << <WeightSum, 1 >> > (weight, WeightSum);
	DelwNull << < WeightSum, 1 >> > (delw, WeightSum);

	cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\0\\1 (1).png");
	/*cv::imshow("Out", image);
	cv::waitKey(10000);*/
	cv::Mat result(image.rows, image.cols, CV_8UC1);
	Input(n, layer, outO, Oout, image);
	t1 = clock();
	for (int adm = 0; adm < 1; adm++) {
		std::cout << "Iter #" << adm << std::endl;
		for (int num = 0; num < 50; num++) {
			for (int k = 0; k < 100; k++) {
				//cv::Mat image = cv::imread("E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num + 1) + ").png");
				//std::cout << "E:\\Foton\\ngnl_data\\training\\" + std::to_string(k) + "\\1 (" + std::to_string(num + 1) + ").png";
				//OutputData(n, layer, outO, Oout, image, result, InputDataArr, Inp, out);
				//outO[k] = 1;
				//hipMemcpy(Oout, outO, n[layer - 1] * sizeof(float), hipMemcpyHostToDevice);

				//NumberInp(out, image, InputDataArr, n, layer, Inp);
				Iteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
				//NumberOut(out, weights, n, layer, NeuralSum);
				Out(NeuralSum, layer, n, weights, out, result);
				//outO[k] = 0;
				//std::cout << "Iter #" << adm << std::endl;
			}
		}
	}

	std::cout << "Time " << clock() - t1 << std::endl;

	/*for (int i = 0; i < 100; i++) {
		int Wnum = 0, Onum = 0, Dnum = 0;
		//std::cin >> filename;
		cv::Mat image = cv::imread("E:\\2.png");
		NumberInp(out, image, InputDataArr, n, layer, Inp);
		cv::imshow("Out", image);
		//Iteration(n, layer, NeuralSum, WeightSum, weight, out, delw, Oout, outO, del);
		for (int i = 0; i < (layer - 1); i++) {
			Sumfunc << <n[i + 1], 1 >> > (n[i], Wnum, Onum, weight, out, n[i + 1]);
			Wnum = Wnum + n[i] * n[i + 1];
			Onum = Onum + n[i];
		}

		NumberOut(out, weights, n, layer, NeuralSum);
	}*/
}
